#include "hip/hip_runtime.h"
#include "cu_cr_solver.h"
#include "cu_cr_internal.h"
#include "cu_cr_functors.cu"

#include <math.h>

/**
* Main method to call in order to solve a tridiagonal matrix using Cyclic-Reduction
*
* Params:
*	n - size of diagonals
* 	vect_* - see diagrams
**/
thrust::host_vector<double>  crSolve(int n, thrust::host_vector<double> vect_a, thrust::host_vector<double> vect_b, thrust::host_vector<double> vect_c, thrust::host_vector<double> vect_d){

	n--; //Cause vectors start at 0
	
	int q = calc_q(n); //Max reduction level
}


/**
* Method used to solve for q when:
* n = 2^q when n is even and
* n = 2^q-1 when n is odd
**/
int calc_q(int n_){
	double n = n_;
	int q = log2(n);
	/*
	if(n_%2==0){
		//q = log2(n);
	} else{
		//q = log2(n)-1;
	}
	*/
	return q;
}


/**
* Method used to calculate the first reduction iteration as it is different
**/
void calc_init(int n,
	thrust::device_ptr<double> d_ptr_a, 
	thrust::device_ptr<double> d_ptr_b, 
	thrust::device_ptr<double> d_ptr_c,
	thrust::device_ptr<double> d_ptr_d){
	
	thrust::device_vector<double> d_vect_alpha(n-1);
	thrust::device_vector<double> d_vect_a_prime(n-1);

	thrust::transform(
		d_ptr_a + 1, d_ptr_a + n,
		d_ptr_b, 
		d_vect_alpha.begin(),
		AlphaBeta()
	);

	thrust::transform(
		d_vect_alpha.begin(),d_vect_alpha.end(),	
		d_ptr_a,
		d_vect_a_prime.begin(),
		thrust::multiplies<double>()
	);
}


