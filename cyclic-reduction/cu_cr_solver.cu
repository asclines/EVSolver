#include "hip/hip_runtime.h"
#include <cyclic-reduction/cu_cr_solver.h>
#include <cyclic-reduction/cu_cr_internal.h>
#include <cyclic-reduction/cu_cr_functors.cu>

#include <hip/hip_runtime.h>
#include <math.h>

#include <thrust/copy.h>
#include <thrust/functional.h>

/*
* For method documentation see cu_cr_internal.h unless otherwise specified.
*/


namespace cyclic_reduction{


void LowerAlphaBeta(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime, DPtrD d_ptr_b){

	InitDPtrD(n,d_ptr_a_prime);
	thrust::transform(
		d_ptr_a + level, d_ptr_a + n,
		d_ptr_b,
		d_ptr_a_prime + level,
		AlphaBeta()
	);
		
}

void UpperAlphaBeta(int n, int level, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	InitDPtrD(n,d_ptr_c_prime);	
	thrust::transform(
		d_ptr_c , d_ptr_c + (n-level),
		d_ptr_b + level,
		d_ptr_c_prime,
		AlphaBeta()
	);

}

//(rank - span >= 0)
void MainFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_b, DPtrD d_ptr_c){

	DVectorD d_vect_temp(n); //TODO see about freeing this memory, and condensing space
	InitDPtrD(n, d_vect_temp.data());
	
	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_c,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b + level, d_ptr_b + n,
		d_vect_temp.begin(),
		d_ptr_b + level,
		thrust::plus<double>()
	);

}


void SolutionFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_d, DPtrD d_ptr_x ){
	DVectorD d_vect_temp(n);

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_d + level,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x + level, d_ptr_x + n,
		d_vect_temp.begin(),
		d_ptr_x + level,
		thrust::plus<double>()
	);

}


void LowerFront(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_a_prime + level,
		d_ptr_a,
		thrust::multiplies<double>()
	);	
}



//(rank + span < n)
void MainBack(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_c_prime, DPtrD d_ptr_b){

	DVectorD d_vect_temp(n);
	InitDPtrD(n, d_vect_temp.data());
	
	thrust::transform(
		d_ptr_c_prime , d_ptr_c_prime + (n - level),
		d_ptr_a + level,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b , d_ptr_b + (n - level),
		d_vect_temp.begin(),
		d_ptr_b,
		thrust::plus<double>()
	);
}

void SolutionBack(int n, int level, DPtrD d_ptr_c_prime, DPtrD d_ptr_d, DPtrD d_ptr_x){
	DVectorD d_vect_temp(n); 
	
	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_d + level,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x + level, d_ptr_x + n,
		d_vect_temp.begin(),
		d_ptr_x + level,
		thrust::plus<double>()
	);

}


void UpperBack(){
//TODO
}



/*
*	Utility Methods
*/

void InitDPtrD(int n, DPtrD d_ptr){
	thrust::fill(
		d_ptr, d_ptr + n,
		0.00
	);				
}

void InitSolutionDPtrD(int n, DPtrD d_ptr_d, DPtrD d_ptr_x){
	thrust::copy_n(d_ptr_d, n, d_ptr_x);	
}

}//END - namespace

/**
* Main method to call in order to solve a tridiagonal matrix using Cyclic-Reduction
*
* Params:
*	n - size of diagonals
* 	vect_* - see diagrams
**/
thrust::host_vector<double>  crSolve(int n, thrust::host_vector<double> vect_a, thrust::host_vector<double> vect_b, thrust::host_vector<double> vect_c, thrust::host_vector<double> vect_d){

	n--; //Cause vectors start at 0
	
	int q = calc_q(n); //Max reduction level
}


/**
* Method used to solve for q when:
* n = 2^q when n is even and
* n = 2^q-1 when n is odd
**/
int calc_q(int n_){
	double n = n_;
	int q = log2(n);
	/*
	if(n_%2==0){
		//q = log2(n);
	} else{
		//q = log2(n)-1;
	}
	*/
	return q;
}


/**
* Method used to calculate the first reduction iteration as it is different
**/
void calc_init(int n,
	thrust::device_ptr<double> d_ptr_a, 
	thrust::device_ptr<double> d_ptr_b, 
	thrust::device_ptr<double> d_ptr_c,
	thrust::device_ptr<double> d_ptr_d){
	
	thrust::device_vector<double> d_vect_alpha(n-1);
	thrust::device_vector<double> d_vect_a_prime(n-1);

	thrust::transform(
		d_ptr_a + 1, d_ptr_a + n,
		d_ptr_b, 
		d_vect_alpha.begin(),
		cyclic_reduction::AlphaBeta()
	);

	thrust::transform(
		d_vect_alpha.begin(),d_vect_alpha.end(),	
		d_ptr_a,
		d_vect_a_prime.begin(),
		thrust::multiplies<double>()
	);
}


