#include "hip/hip_runtime.h"
#include <cyclic-reduction/cu_cr_solver.h>
#include <cyclic-reduction/cu_cr_internal.h>
#include <cyclic-reduction/cu_cr_functors.cu>

#include <hip/hip_runtime.h>
#include <math.h>
#include <thread>

#include <thrust/copy.h>
#include <thrust/functional.h>

/*
* For method documentation see cu_cr_internal.h unless otherwise specified.
*/


namespace cyclic_reduction{

HVectorD Solve(int size, HVectorD h_vect_a, HVectorD h_vect_b, HVectorD h_vect_c, HVectorD h_vect_d){

	DVectorD d_vect_a,
		d_vect_b,
		d_vect_c,
		d_vect_d,
		d_vect_x(size,0.00),
		d_vect_a_prime(size,0.00),
		d_vect_c_prime(size,0.00);


	d_vect_a = h_vect_a;
	d_vect_b = h_vect_b;
	d_vect_c = h_vect_c;
	d_vect_d = h_vect_d;

	

//Foward Reduction Phase

	int level = 1;
	while(level < size){

	//AlphaBeta Methods
/*		std::thread lab(LowerAlphaBeta,
				size,level,
				d_vect_a.data(),
				d_vect_a_prime.data(),
				d_vect_b.data()
		);

		std::thread uab(UpperAlphaBeta,
				size,level,
				d_vect_b.data(),
				d_vect_c.data(),
				d_vect_c_prime.data()
		);
*/
		d_vect_x = d_vect_d;

//		lab.join();
//		uab.join();

		LowerAlphaBeta(size,level,
			d_vect_a.data(),
			d_vect_a_prime.data(),
			d_vect_b.data()
		);

		UpperAlphaBeta(size, level,
			d_vect_b.data(),
			d_vect_c.data(),
			d_vect_c_prime.data()
		);
	
	
	//Front Methods
		
		MainFront(size, level,
			d_vect_a_prime.data(),
			d_vect_b.data(),
			d_vect_c.data()
		);

		SolutionFront(size, level,
			d_vect_a_prime.data(),
			d_vect_d.data(),
			d_vect_x.data()
		);

		LowerFront(size, level,
			d_vect_a.data(),
			d_vect_a_prime.data()
		);

	//Back Methods

		MainBack(size, level,
			d_vect_a.data(),
			d_vect_c_prime.data(),
			d_vect_b.data()
		);

		SolutionBack(size, level,
			d_vect_c_prime.data(),
			d_vect_d.data(),
			d_vect_x.data()
		);

		UpperBack(size, level,
			d_vect_c.data(),
			d_vect_c_prime.data()
		);			

	//Set up diagonals for next reduction level
		d_vect_a = d_vect_a_prime;
		d_vect_c = d_vect_c_prime;
		d_vect_d = d_vect_x;

		level *= 2;
	}

//Backward Substitution Phase
	DVectorD d_vect_results(size);
		thrust::transform(
			d_vect_d.begin(), d_vect_d.end(),
			d_vect_b.begin(),
			d_vect_results.begin(),
			thrust::divides<double>()
		);

	h_vect_d = d_vect_results;
	
				
	return h_vect_d;
}


void LowerAlphaBeta(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime, DPtrD d_ptr_b){

//	InitDPtrD(n,d_ptr_a_prime);
	thrust::transform(
		d_ptr_a + level, d_ptr_a + n,
		d_ptr_b,
		d_ptr_a_prime + level,
		AlphaBeta()
	);
		
}

void UpperAlphaBeta(int n, int level, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

//	InitDPtrD(n,d_ptr_c_prime);	
	thrust::transform(
		d_ptr_c , d_ptr_c + (n-level),
		d_ptr_b + level,
		d_ptr_c_prime,
		AlphaBeta()
	);

}

//(rank - span >= 0)
void MainFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_b, DPtrD d_ptr_c){

	DVectorD d_vect_temp(n); //TODO see about freeing this memory, and condensing space
	InitDPtrD(n-level, d_vect_temp.data());
	
	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_c,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b + level, d_ptr_b + n,
		d_vect_temp.begin(),
		d_ptr_b + level,
		thrust::plus<double>()
	);

}


void SolutionFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_d, DPtrD d_ptr_x ){
	DVectorD d_vect_temp(n-level);

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_d,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x + level, d_ptr_x + n,
		d_vect_temp.begin(),
		d_ptr_x + level,
		thrust::plus<double>()
	);

}


void LowerFront(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_a,
		d_ptr_a_prime + level,
		thrust::multiplies<double>()
	);	
}



//(rank + span < n)
void MainBack(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_c_prime, DPtrD d_ptr_b){

	DVectorD d_vect_temp(n-1,0.00);
//	InitDPtrD(n, d_vect_temp.data());
	
	thrust::transform(
		d_ptr_c_prime , d_ptr_c_prime + (n - level),
		d_ptr_a + level,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b , d_ptr_b + (n - level),
		d_vect_temp.begin(),
		d_ptr_b,
		thrust::plus<double>()
	);
}

void SolutionBack(int n, int level, DPtrD d_ptr_c_prime, DPtrD d_ptr_d, DPtrD d_ptr_x){
	DVectorD d_vect_temp(n-level); 
	
	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_d + level,
		d_vect_temp.begin(),
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x , d_ptr_x + (n-level),
		d_vect_temp.begin(),
		d_ptr_x,
		thrust::plus<double>()
	);

}


void UpperBack(int n, int level, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_c + level,
		d_ptr_c_prime,
		thrust::multiplies<double>()
	);	
}



/*
*	Utility Methods
*/

void InitDPtrD(int n, DPtrD d_ptr){
	thrust::fill(
		d_ptr, d_ptr + n,
		0.00
	);				
}

void InitSolutionDPtrD(int n, DPtrD d_ptr_d, DPtrD d_ptr_x){
	thrust::copy_n(d_ptr_d, n, d_ptr_x);	
}

}//END - namespace



/**
* Method used to solve for q when:
* n = 2^q when n is even and
* n = 2^q-1 when n is odd
**/
int calc_q(int n_){
	double n = n_;
	int q = log2(n);
	/*
	if(n_%2==0){
		//q = log2(n);
	} else{
		//q = log2(n)-1;
	}
	*/
	return q;
}


/**
* Method used to calculate the first reduction iteration as it is different
**/
void calc_init(int n,
	thrust::device_ptr<double> d_ptr_a, 
	thrust::device_ptr<double> d_ptr_b, 
	thrust::device_ptr<double> d_ptr_c,
	thrust::device_ptr<double> d_ptr_d){
	
	thrust::device_vector<double> d_vect_alpha(n-1);
	thrust::device_vector<double> d_vect_a_prime(n-1);

	thrust::transform(
		d_ptr_a + 1, d_ptr_a + n,
		d_ptr_b, 
		d_vect_alpha.begin(),
		cyclic_reduction::AlphaBeta()
	);

	thrust::transform(
		d_vect_alpha.begin(),d_vect_alpha.end(),	
		d_ptr_a,
		d_vect_a_prime.begin(),
		thrust::multiplies<double>()
	);
}


