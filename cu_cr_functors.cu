#include <hip/hip_runtime.h>


/**
* Calculates:
*	alpha_i = -a_i/b_(i-2^(l-i))
*	where:
*		x = a_i
*		y = b_(i-2^(l-i))
*
*	beta_i = -c_i/b_(i+2^(l-i))
*	where:
*		x = c_i
*		y = b_(i+2^(l-i))	
**/
struct AlphaBeta{
	__host__ __device__
	double operator()(double x, double y){
		return (-x)/y;
	}
};


