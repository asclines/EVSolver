#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include <serial_tSolver.h>

TEST( serialTest, systemTest )
{
	double n = 4;

	host_vector<double> a(4);
	a[0] = 0.0; a[1] = -1.0; a[2] = -1.0; a[3] = -1.0;
	host_vector<double> b(4);
	b[0] = 4.0; b[1] = 4.0; b[2] = 4.0; b[3] = 4.0;
	host_vector<double> c(4);
	c[0] = -1.0; c[1] = -1.0; c[2] = -1.0; c[3] = 0.0;
	host_vector<double> d(4);
	d[0] = 5; d[1] = 5; d[2] = 10; d[3] = 23.0;

	host_vector<double> actual(4);
	host_vector<double> exp(4);
	exp[0] = 2; exp[1] = 3; exp[2] = 5; exp[7];

	actual = serial_solve(a, b, c, d);


	for(int i = 0; i < n; i++)
	{
		EXPECT_EQ(exp[i], actual[i]);
	}
}
